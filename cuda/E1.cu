#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgb_grey(const uchar4* const rgbaImage,unsigned char* const greyImage,int nr, int nc)
{
      int indx.x = threadIdx.x;  
      int indx.y = threadIdx.y;
      int bindx.x = blockIdx.x;
      int bindx.y = blockIdx.y;
      
      int bdim.x = blockDim.x;
      int bdim.y = blockDim.y; 
      int gdim.x = gridDim.x;
      int gdim.y = gridDim.y;
      
      int xp = bdim.x * bindx.x + indx.x;
      int yp = bdim.y * bindx.y + indx.y;
          
      int offset =  yp * (bdim.x * gdim.x) + xp;
      
      uchar4 rgb = rgbaImage[offset];
      float chSum = .299f * rgb.x + .587f * rgb.y + .114f * rgb.z;
      greyImage[offset] = chSum; 
    
}

void rgb_grey1(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,unsigned char* const d_greyImage, size_t nr, size_t nc)
{
  
  const dim3 blockSize(nr/16+1, nc/16+1, 1);  //TODO
  const dim3 gridSize( 16, 16, 1);  //TODO
  rgb_grey<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, nr, nc);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}