#include "hip/hip_runtime.h"
#include "utils.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <>
#include <hip/device_functions.h>
#include <thrust/sort.h>
__global__ void print_kernel(unsigned int *d_out)
{
  printf("%d ", d_out[threadIdx.x]);
}


__global__ void histo_kernel(unsigned int * d_out, unsigned int* const d_in,unsigned int shift, const unsigned int numElems)
{
  unsigned int mask = 1 << shift;
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  if (myId >= numElems)  return;
  int bin = (d_in[myId] & mask) >> shift;
  atomicAdd(&d_out[bin], 1);
}

__global__ void sumscan_kernel(unsigned int * d_in, const size_t numBins, const unsigned int numElems)
{
  int myId = threadIdx.x;
  if (myId >= numElems)  return;
  extern __shared__ float sdata[];
  sdata[myId] = d_in[myId];
  __syncthreads();            // make sure entire block is loaded!

  for (int d = 1; d < numBins; d *= 2) {
    if (myId >= d) {
      sdata[myId] += sdata[myId - d];
    }
    __syncthreads();
  }
  if (myId == 0)  d_in[0] = 0;
  else  d_in[myId] = sdata[myId - 1]; //inclusive->exclusive
}

__global__ void makescan_kernel(unsigned int * d_in, unsigned int *d_scan,unsigned int shift, const unsigned int numElems)
{
  unsigned int mask = 1 << shift;
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  if (myId >= numElems)  return;
  d_scan[myId] = ((d_in[myId] & mask) >> shift) ? 0 : 1;
}

__global__ void move_kernel(unsigned int* const d_inputVals,
  unsigned int* const d_inputPos,
  unsigned int* const d_outputVals,
  unsigned int* const d_outputPos,
  const unsigned int numElems,
  unsigned int* const d_histogram,
  unsigned int* const d_scaned,
  unsigned int shift)
{
  unsigned int mask = 1 << shift;
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  if (myId >= numElems)  return;
  int des_id = 0;
  if ((d_inputVals[myId] & mask) >> shift) {
    des_id = myId + d_histogram[1] - d_scaned[myId];
  } else {
    des_id = d_scaned[myId];
  }
  d_outputVals[des_id] = d_inputVals[myId];
  d_outputPos[des_id] = d_inputPos[myId];
}

#ifdef USE_THRUST
void your_sort(unsigned int* const d_inputVals,unsigned int* const d_inputPos,unsigned int* const d_outputVals,unsigned int* const d_outputPos,const size_t numElems)
{
  thrust::device_ptr<unsigned int> d_inputVals_p(d_inputVals);
  thrust::device_ptr<unsigned int> d_inputPos_p(d_inputPos);
  thrust::host_vector<unsigned int> h_inputVals_vec(d_inputVals_p,d_inputVals_p + numElems);
  thrust::host_vector<unsigned int> h_inputPos_vec(d_inputPos_p,d_inputPos_p + numElems);
  thrust::sort_by_key(h_inputVals_vec.begin(), h_inputVals_vec.end(), h_inputPos_vec.begin());
  checkCudaErrors(hipMemcpy(d_outputVals, thrust::raw_pointer_cast(&h_inputVals_vec[0]),numElems * sizeof(unsigned int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_outputPos, thrust::raw_pointer_cast(&h_inputPos_vec[0]),numElems * sizeof(unsigned int), hipMemcpyHostToDevice));
}
#else
void your_sort(unsigned int* const d_inputVals,
  unsigned int* const d_inputPos,
  unsigned int* const d_outputVals,
  unsigned int* const d_outputPos,
  const size_t numElems)
{
  const int numBits = 1;  //??
  const int numBins = 1 << numBits;
  const int m = 1 << 10;
  int blocks = ceil((float)numElems / m);
  printf("m %d blocks %d\n", m ,blocks);
  unsigned int *d_binHistogram;
  checkCudaErrors(hipMalloc(&d_binHistogram, sizeof(unsigned int)* numBins));
  thrust::device_vector<unsigned int> d_scan(numElems);
  for (unsigned int i = 0; i < 8 * sizeof(unsigned int); i++) {
    checkCudaErrors(hipMemset(d_binHistogram, 0, sizeof(unsigned int)* numBins));
    histo_kernel << <blocks, m >> >(d_binHistogram, d_inputVals, i, numElems);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    sumscan_kernel << <1, numBins, sizeof(unsigned int)* numBins>> >(d_binHistogram, numBins, numElems);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    makescan_kernel << <blocks, m >> >(d_inputVals, thrust::raw_pointer_cast(&d_scan[0]), i, numElems);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    thrust::exclusive_scan(d_scan.begin(), d_scan.end(), d_scan.begin());
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    move_kernel << <blocks, m >> >(d_inputVals, d_inputPos, d_outputVals, d_outputPos,numElems, d_binHistogram, thrust::raw_pointer_cast(&d_scan[0]), i);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipMemcpy(d_inputVals, d_outputVals, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(d_inputPos, d_outputPos, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice));
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
  }
  checkCudaErrors(hipFree(d_binHistogram));
}
#endif