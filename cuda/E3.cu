#include "hip/hip_runtime.h"
#include "utils.h"
#include<>
#include<hip/device_functions.h>

__global__ void shmem_reduce_kernel(float * d_out, const float * const d_in, bool is_max)
{
  extern __shared__ float sdata[];

  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int tid = threadIdx.x;
  sdata[tid] = d_in[myId];
  __syncthreads();            // make sure entire block is loaded!
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      if (is_max)
        sdata[tid] = max(sdata[tid], sdata[tid + s]);
      else
        sdata[tid] = min(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }
  if (tid == 0)
  {
    d_out[blockIdx.x] = sdata[0];
  }
}

__global__ void histo_kernel(unsigned int * d_out, const float * const d_in,
  const size_t numBins, float logLumRange, float min_logLum)
{
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int bin = (d_in[myId] - min_logLum) / logLumRange * numBins;
  if (bin == numBins)  bin--;
  atomicAdd(&d_out[bin], 1);
}

__global__ void scan_kernel(unsigned int * d_out, const float * const d_in,
  const size_t numBins, float logLumRange, float min_logLum)
{
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int bin = (d_in[myId] - min_logLum) / logLumRange * numBins;
  if (bin == numBins)  bin--;
  atomicAdd(&d_out[bin], 1);
}
__global__ void cdf_kernel(unsigned int * d_in, const size_t numBins)
{
  int myId = threadIdx.x;
  for (int d = 1; d < numBins; d *= 2) {
    if ((myId + 1) % (d * 2) == 0) {
      d_in[myId] += d_in[myId - d];
    }
    __syncthreads();
  }
  if (myId == numBins - 1) d_in[myId] = 0;
  for (int d = numBins / 2; d >= 1; d /= 2) {
    if ((myId + 1) % (d * 2) == 0) {
      unsigned int tmp = d_in[myId - d];
      d_in[myId - d] = d_in[myId];
      d_in[myId] += tmp;
    }
    __syncthreads();
  }
}
__global__ void cdf_kernel_2(unsigned int * d_in, const size_t numBins)
{ 
  int idx = threadIdx.x;
  extern __shared__ int temp[];
  int pout = 0, pin = 1;

  temp[idx] = (idx > 0) ? d_in[idx - 1] : 0;
  __syncthreads();

  for (int offset = 1; offset < n; offset *= 2) {
    pout = 1 - pout;
    pin = 1 - pout;
    if (idx >= offset) {
      temp[pout*n+idx] = temp[pin*n+idx - offset] + temp[pin*n+idx];  // changed line
    } else {
      temp[pout*n+idx] = temp[pin*n+idx];
    }
    __syncthreads();
  }
  d_in[idx] = temp[pout*n+idx];
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
  unsigned int* const d_cdf,
  float &min_logLum,
  float &max_logLum,
  const size_t numRows,
  const size_t numCols,
  const size_t numBins)
{
  const int m = 1 << 10;
  int blocks = ceil((float)numCols * numRows / m);

  float *d_intermediate; // should not modify d_in
  checkCudaErrors(hipMalloc(&d_intermediate, sizeof(float)* blocks)); // store max and min
  float *d_min, *d_max;
  checkCudaErrors(hipMalloc((void **)&d_min, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_max, sizeof(float)));

  shmem_reduce_kernel << <blocks, m, m * sizeof(float) >> >(d_intermediate, d_logLuminance, true);
  shmem_reduce_kernel << <1, blocks, blocks * sizeof(float) >> >(d_max, d_intermediate, true);
  shmem_reduce_kernel << <blocks, m, m * sizeof(float) >> >(d_intermediate, d_logLuminance, false);
  shmem_reduce_kernel << <1, blocks, blocks * sizeof(float) >> >(d_min, d_intermediate, false);
  checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_intermediate));
  checkCudaErrors(hipFree(d_min));
  checkCudaErrors(hipFree(d_max));
  float logLumRange = max_logLum - min_logLum;
  printf("max_logLum: %f  min_logLum: %f  logLumRange: %f\n", max_logLum, min_logLum, logLumRange);
  checkCudaErrors(hipMemset(d_cdf, 0, sizeof(unsigned int)* numBins));
  histo_kernel << <blocks, m >> >(d_cdf, d_logLuminance, numBins, logLumRange, min_logLum);
  cdf_kernel_2 << <1, numBins, sizeof(unsigned int) * numBins * 2 >> >(d_cdf, numBins);
}