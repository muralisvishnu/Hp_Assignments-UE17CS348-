#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"

__global__
void gb(const unsigned char* const ic,unsigned char* const oc,int nr, int nc,const float* const filter, const int filterWidth)                                      
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * nc + thread_2D_pos.x;
  if (thread_2D_pos.x >= nc || thread_2D_pos.y >= nr)
    return;
  float result = 0.f;
  for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
    for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {
      int image_r = min(max(thread_2D_pos.y + filter_r, 0), static_cast<int>(nr - 1));
      int image_c = min(max(thread_2D_pos.x + filter_c, 0), static_cast<int>(nc - 1));

      float image_value = static_cast<float>(ic[image_r * nc + image_c]);
      float filter_value = filter[(filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2];

      result += image_value * filter_value;
    }
  }
  oc[thread_1D_pos] = result;
}
__global__
void sc(const uchar4* const iprgb,int nr,int nc,unsigned char* const rc,unsigned char* const gc,unsigned char* const bc)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * nc + thread_2D_pos.x;
  if (thread_2D_pos.x >= nc || thread_2D_pos.y >= nr)
    return;rc[thread_1D_pos] = iprgb[thread_1D_pos].x;gc[thread_1D_pos] = iprgb[thread_1D_pos].y;bc[thread_1D_pos] = iprgb[thread_1D_pos].z;
}
__global__
void recombineChannels(const unsigned char* const rc,const unsigned char* const gc,const unsigned char* const bc,uchar4* const oprgb,int nr,int nc)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,blockIdx.y * blockDim.y + threadIdx.y);
  const int thread_1D_pos = thread_2D_pos.y * nc + thread_2D_pos.x;
  if (thread_2D_pos.x >= nc || thread_2D_pos.y >= nr)
    return;
  unsigned char red   = rc[thread_1D_pos];
  unsigned char green = gc[thread_1D_pos];
  unsigned char blue  = bc[thread_1D_pos];
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);
  oprgb[thread_1D_pos] = outputPixel;
}
unsigned char *d_red, *d_green, *d_blue;
float *d_filter;
void allocateMemoryAndCopyToGPU(const size_t nrImage, const size_t ncImage,const float* const h_filter, const size_t filterWidth)
{
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * nrImage * ncImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * nrImage * ncImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * nrImage * ncImage));
  int num_filter_bytes = sizeof(float) * filterWidth * filterWidth;
  checkCudaErrors(hipMalloc(&d_filter, num_filter_bytes));
  checkCudaErrors(hipMemcpy(d_filter, h_filter, num_filter_bytes, hipMemcpyHostToDevice));
}
void your_gb(const uchar4 * const h_iprgb, uchar4 * const d_iprgb,uchar4* const d_oprgb, const size_t nr, const size_t nc,unsigned char *d_redBlurred, unsigned char *d_greenBlurred, unsigned char *d_blueBlurred,const int filterWidth)
{
  const dim3 blockSize(1, 1, 1);
  const dim3 gridSize(nc, nr, 1);
  sc<<<gridSize, blockSize>>>(d_iprgb,nr,nc,d_red,d_green,d_blue);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gb<<<gridSize, blockSize>>>(d_red,d_redBlurred,nr,nc,d_filter,filterWidth);
  gb<<<gridSize, blockSize>>>(d_green,d_greenBlurred,nr,nc,d_filter,filterWidth);
  gb<<<gridSize, blockSize>>>(d_blue,d_blueBlurred,nr,nc,d_filter,filterWidth);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,d_greenBlurred,d_blueBlurred,d_oprgb,nr,nc);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}